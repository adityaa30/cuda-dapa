#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>

#define DEBUG false
#define NULL_DEFAULT -1

__device__ int x[] = {0, -1, 0, 1};
__device__ int y[] = {-1, 0, 1, 0};

__host__ void PrintMatrix(int *Mat, int n, bool transpose) {
  if (transpose)
    printf("Transposed Matrix:\n");
  else
    printf("Original Matrix:\n");

  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < n; ++j) {
      printf("%d\t", Mat[i * n + j]);
    }
    printf("\n");
  }
}

/**
1. A(i, j) is used to store aij initially and aji when the algorithm terminates;
2. B(i, j) is used to store data received from P(i, j + 1) or P(i - 1, j), that
is, from its right or top neighbors; and
3. C(i, j) is used to store data received from P(i, j - 1) or P(i + 1, j), that
is, from its left or bottom neighbors.
*/

typedef struct {
  int a_km = NULL_DEFAULT, m = -1, k = -1;
  bool isNull = true;
} Data;

__device__ void CopyData(Data &dest, Data &src) {
  dest.a_km = src.a_km;
  dest.m = src.m;
  dest.k = src.k;
  dest.isNull = false;
}

__device__ void PrintData(Data &data) {
  printf("(%d, %d, %d %d)\t", data.a_km, data.m, data.k, data.isNull ? 1 : 0);
}

__device__ void MakeDataNull(Data &data) {
  data.isNull = true;
  data.m = -1;
  data.k = -1;
  data.a_km = NULL_DEFAULT;
}

__device__ void PrintDebugMatrix(int *A, Data *B, Data *C, int n, float step) {
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    printf("Debug matrix: Step #%1.1f\n", step);
    for (int i = 0; i < n; ++i) {
      for (int j = 0; j < n; ++j) {
        int idx = i * n + j;
        int val_b = B[idx].isNull ? NULL_DEFAULT : B[idx].a_km;
        int val_c = C[idx].isNull ? NULL_DEFAULT : C[idx].a_km;
        printf("(%d, %d, %d)\t", A[idx], val_b, val_c);
      }
      printf("\n");
    }
  }
}

__device__ bool isValid(int x, int y, int n) {
  return (0 <= x && x < n && 0 <= y && y < n);
}

__global__ void MeshTranspose(int *A, Data *B, Data *C, int n) {
  // printf("[thread (%d, %d)] Start.\n", threadIdx.x, threadIdx.y);

  int i = blockDim.x * blockIdx.x + threadIdx.x;
  int j = blockDim.y * blockIdx.y + threadIdx.y;
	
	if(i >= n || j >= n) return;

	int i_j = i * n + j;

  MakeDataNull(B[i_j]);
  MakeDataNull(C[i_j]);

  // Step 1.1
  if (1 <= i && i < n && 0 <= j && j < i) {
    C[(i - 1) * n + j] = {A[i_j], j, i, false};
  }

  // Step 1.2
  if (0 <= i && i < n - 1 && i < j && j < n) {
    B[i * n + (j - 1)] = {A[i_j], j, i, false};
  }

  if (DEBUG)
    PrintDebugMatrix(A, B, C, n, 1.0f);

  for (int step = 0; step <= (2 * n + 3); ++step) {
    // printf("[thread (%d, %d)] Step %d.\n", threadIdx.x, threadIdx.y, step);

    // Declare local variables to implement message passing
    int temp_A = NULL_DEFAULT;
    bool got_A_from_B = false;
    Data *temp_B = new Data[4];
    Data *temp_C = new Data[4];

    // Step 2.1
    if (1 <= i && i < n && 0 <= j && j < i) {
      // (a_km, m, k) is received from P(i + 1, j)
      // send it to P(i - 1, j)
      CopyData(temp_C[1], C[i_j]);
      // (a_km, m, k) is received from P(i - 1, j)
      if (B[i_j].m == i && B[i_j].k == j) {
        // A(i, j) <- a_km {a_km has reached its destination}
        temp_A = B[i_j].a_km;
        got_A_from_B = true;
      } else {
        // Send (a_km, m, k) to P(i + 1, j)
        CopyData(temp_B[3], B[i_j]);
      }
    }

    // Step 2.2
    int i_i = i * n + i;
    if (0 <= i && i < n && i == j) {
      // (a_km, m, k) is received from P(i + 1, i)
      // Send it to P(i, i + 1)
      CopyData(temp_C[2], C[i_i]);

      // (a_km, m, k) is received from P(i, i + 1)
      // Send it to P(i + 1, i)
      CopyData(temp_B[3], B[i_i]);
    }

    // Step 2.3
    if (0 <= i && i < n - 1 && i < j && j < n) {
      // (a_km, m, k) is received from P(i, j + 1)
      // send it to P(i, j - 1)
      CopyData(temp_B[0], B[i_j]);
      // (a_km, m, k) is received from P(i, j - 1)
      if (C[i_j].m == i && C[i_j].k == j) {
        // A(i, j) <- a_km {a_km has reached its destination}
        temp_A = C[i_j].a_km;
        got_A_from_B = false;
      } else {
        // Send (a_km, m, k) to P(i, j + 1)
        CopyData(temp_C[2], C[i_j]);
      }
    }

		__syncthreads();
    // Copy the final state values now
    // Below section only deals with writing of data
    if (temp_A != NULL_DEFAULT) {
      A[i_j] = temp_A;
      if (got_A_from_B)
        MakeDataNull(B[i_j]);
      else
        MakeDataNull(C[i_j]);
    }
    for (int next = 0; next < 4; ++next) {
      int new_x = i + x[next];
      int new_y = j + y[next];
      int idx = new_x * n + new_y;
      if (isValid(new_x, new_y, n)) {
        if (temp_B[next].isNull == false) {
          CopyData(B[idx], temp_B[next]);
        }
        if (temp_C[next].isNull == false) {
          CopyData(C[idx], temp_C[next]);
        }
      }
    }

    // Now make null for the last row in C 2D-Array
    if (i == n - 1) {
      MakeDataNull(C[i_j]);
    }

    // Now make null for the last column in B 2D-Array
    if (j == n - 1) {
      MakeDataNull(B[i_j]);
    }

    if (DEBUG)
      PrintDebugMatrix(A, B, C, n, 2.0f + step / 10.0f);
  }
}

int main() {
  int n = 20;
  printf("n=%d\n", n);

  int *Mat = new int[n * n];
  int val = 0;
  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < n; ++j) {
      Mat[i * n + j] = ++val;
    }
  }

  int *d_A;
  Data *d_B, *d_C;
  hipMalloc((void **)&d_A, sizeof(int) * (n * n));
  hipMalloc((void **)&d_B, sizeof(Data) * (n * n));
  hipMalloc((void **)&d_C, sizeof(Data) * (n * n));
  hipMemcpy(d_A, Mat, sizeof(int) * (n * n), hipMemcpyHostToDevice);

  dim3 block_size(32, 32);
  dim3 grid_size(1);

  if (DEBUG)
    PrintMatrix(Mat, n, 0);

  clock_t time_taken = clock();
  MeshTranspose<<<grid_size, block_size>>>(d_A, d_B, d_C, n);
  hipMemcpy(Mat, d_A, sizeof(int) * (n * n), hipMemcpyDeviceToHost);
  time_taken = clock() - time_taken;

  printf("Transpose Done.\n");

  if (DEBUG)
    PrintMatrix(Mat, n, 1);
  printf("Time taken: %f ms\n", 1000 * ((float)time_taken / CLOCKS_PER_SEC));

	// Testing if output is correct!
	bool ok = true;
	val = 0;
	for(int j = 0; j < n; ++j) {
		for(int i = 0; i < n; ++i) {
       if(Mat[i *n + j] != (++val)) {
				 ok = false;
			 }
		}
	}

	if(ok) printf("Test: OK\n");
	else {
		printf("Test: FAIL\n");
    PrintMatrix(Mat, n, 1);
	}

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(Mat);
}